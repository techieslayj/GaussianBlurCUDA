#include "hip/hip_runtime.h"
//AJ Iglesias//
//Blur Blur Blur//
//Shared mem//

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "Jpegfile.h"

#define block 32
using namespace std;

//use device for image blur here ints index and r are indexing for pixels by thread and the radius respectively
__global__ void blur(BYTE * devdataBuf, BYTE * devprocessBuf, int width, int height){


	int i = threadIdx.x;
	int j = threadIdx.y;
	int I = blockIdx.x * block + i;
	int J = blockIdx.y * block + j;



	int weights[5][5] = { {1, 4, 6, 4, 1}, {4, 16, 24, 16, 4}, {6, 24, 36, 24, 6}, {4, 16, 24, 16, 4}, {1, 4, 6, 4, 1 }};
	__shared__  BYTE pixels[block+4][block+4][3];


	if(I < width && J < height){
		//this is what we manipulate i's and J's when doing bottom, top, left, right stuff
		pixels[j+2][i+2][0] = devdataBuf[(J * width + I)*3];
		pixels[j+2][i+2][1] = devdataBuf[(J * width + I)*3+ 1];
		pixels[j+2][i+2][2] = devdataBuf[(J * width + I)*3+ 2];
		int left = blockIdx.x * block -2;
		int right = blockIdx.x * block + block-1 + 2;
		int bottom = blockIdx.y* block -2;
		int top = blockIdx.y * block + block-1 +2;

		if(i < 2) {
			if(left > 0){
				pixels[j+2][i][0] = devdataBuf[(J * width + left + i)*3];
				pixels[j+2][i][1] = devdataBuf[(J * width + left + i)*3 + 1];
				pixels[j+2][i][2] = devdataBuf[(J * width + left + i)*3 + 2];

				if( j < 2) {
					if (bottom > 0) {
						pixels[j][i][0] = devdataBuf[((bottom+j)*width+left+i)*3];
						pixels[j][i][1] = devdataBuf[((bottom+j)*width+left+i)*3+ 1];
						pixels[j][i][2] = devdataBuf[((bottom+j)*width+left+i)*3+ 2];
					}
				}
				if(j >= block-2){
					int jj = j - (block-2);
					if(top < height) {
						pixels[block+3-jj][i][0] = devdataBuf[((top-jj)*width+left+i)*3];
						pixels[block+3-jj][i][1] = devdataBuf[((top-jj)*width+left+i)*3+ 1];
						pixels[block+3-jj][i][2] = devdataBuf[((top-jj)*width+left+i)*3+ 2];
					}
			}
		}

	}
		if(i >= block-2){
			//update right ghost
			int ii = i - (block-2);
			if(right < width){
				pixels[j+2][block+3-ii][0] = devdataBuf[(J * width + right - ii)*3];
				pixels[j+2][block+3-ii][1] = devdataBuf[(J * width + right - ii)*3 + 1];
				pixels[j+2][block+3-ii][2] = devdataBuf[(J * width + right - ii)*3 + 2];
				//pixels[j+2][block+3-ii][2] = devdataBuf[(J * width + right - ii)*3 + 2];

				//update right bottom ghosts
				if(j < 2){
					if(bottom > 0){
						pixels[j][block-ii+3][0] = devdataBuf[((bottom+j) * width + right - ii)*3];
						pixels[j][block-ii+3][1] = devdataBuf[((bottom+j) * width + right- ii)*3 + 1];
						pixels[j][block-ii+3][2] = devdataBuf[((bottom+j) * width + right - ii)*3 + 2];
						//pixels[j][i+2][2] = devdataBuf[(J * width + top + block - 2 + i)*3 + 2];
					}

				}


				//update right top ghosts
				if( j >= block-2){
					int jj = j - (block-2);
					if ( top < height){
						pixels[block+3-jj][block-ii+3][0] = devdataBuf[((top-1+jj)* width +right - ii)*3];
						pixels[block+3-jj][block-ii+3][1] = devdataBuf[((top-1+jj)* width +right - ii)*3 + 1];
						pixels[block+3-jj][block-ii+3][2] = devdataBuf[((top-1+jj)* width +right - ii)*3 + 2];

					}
			}
		}
}


		if(j < 2){
			//update bottom ghost
			if(bottom > 0){
				pixels[j][i+2][0] = devdataBuf[((bottom+j) * width + I)*3];
				pixels[j][i+2][1] = devdataBuf[((bottom+j) * width + I)*3 + 1];
				pixels[j][i+2][2] = devdataBuf[((bottom+j) * width + I)*3 - 2];

			}

		}
		if(j >= block-2){
			//update top ghost
			int jj = j - (block-2);
			if(top < height){
				pixels[block+3-jj][i+2][0] = devdataBuf[((top-jj) * width + I)*3];
				pixels[block+3-jj][i+2][1] = devdataBuf[((top-jj) * width + I)*3 + 1];
				pixels[block+3-jj][i+2][2] = devdataBuf[((top-jj) * width + I)*3 + 2];


			}
}
}

	__syncthreads();


	if(I < width && J < height) {
		int r = 0, g = 0, b = 0;
		int totw = 0;

		for(int row = -2; row <= 2; row++){
			for(int col = -2; col <= 2; col++){
				int w = weights[row+2][col+2];
				r += pixels[j+row+2][i+col+2][0] * w;
				g += pixels[j+row+2][i+col+2][1] * w;
				b += pixels[j+row+2][i+col+2][2] * w;
				totw += w;
				//pRed = pixels[nrow * width * 3 + ncol * 3];
				//pGrn = pixels[nrow * width * 3 + ncol * 3 + 1];
				//pBlu = pixels[nrow * width * 3 + ncol * 3 + 2];

				//avgRed += (double)(*pRed);
				//avgGrn += (double)(*pGrn);
				//avgBlu += (double)(*pBlu);
				//pixCount++;
			}
		}

		devprocessBuf[(J*width+I)*3] = (BYTE)(r / totw);
		devprocessBuf[(J*width+I)*3+1] = (BYTE)(g / totw);
		devprocessBuf[(J*width+I)*3+2] = (BYTE)(b / totw);

}
}


int main(int argc, char* argv[]){


	// set up BYTE arrays
	BYTE *dataBuf, *ddataBuf;
	BYTE *processBuf, *dprocessBuf;

	UINT height, width;
	//read the file to dataBuf with RGB format
	//host reads in image i.e. host code

	dataBuf = JpegFile::JpegFileToRGB("MBAPPE.jpg", &width, &height);

	int size = 3 * width * height * sizeof(BYTE);

	hipEvent_t start,stop;

	float cudaElapsed = 0;

	//allocate memory on device buffers

	hipMalloc((void **)&ddataBuf, size * 3);
	hipMalloc((void **)&dprocessBuf, size * 3);


	processBuf = (BYTE*) calloc(height * width * 3, sizeof(BYTE));

	cudaElapsed = 0;

		//copy dataBuf to device
	hipMemcpy(ddataBuf, dataBuf, size, hipMemcpyHostToDevice);

	//establish threads
	dim3 dimgrid((width+block-1)/block,(height+block-1)/block, 1);

	dim3 dimblock(block, block, 1);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	//Run device blur function
	blur<<<dimgrid,dimblock>>>(ddataBuf, dprocessBuf, width, height);



	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&cudaElapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(processBuf, dprocessBuf, size, hipMemcpyDeviceToHost);

	//Print out corresponding threads processing time
	cout << "Processing Time: " << cudaElapsed << endl;


JpegFile::RGBToJpegFile("MBAPPEblur.jpg", processBuf, width, height, true, 75);
hipFree(dprocessBuf);
hipFree(ddataBuf);

return 0;

}
